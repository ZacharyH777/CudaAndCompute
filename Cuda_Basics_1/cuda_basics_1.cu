#include "hip/hip_runtime.h"
#include <iostream>

__global__ void addKernel(int* c, const int* a, const int* b) {
    int i = threadIdx.x; 
    c[i] = a[i] + b[i];
}

__host__ int main() {
    const int arraySize = 5;
    int a[arraySize] = { 1, 2, 3, 4, 5 };
    int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    int* dev_a, * dev_b, * dev_c;

    hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    hipMalloc((void**)&dev_c, arraySize * sizeof(int));

    hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

    addKernel << <1, arraySize >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Result: ";
    for (int i = 0; i < arraySize; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}