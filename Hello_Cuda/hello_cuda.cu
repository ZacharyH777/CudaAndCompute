#include "hip/hip_runtime.h"
#include <iostream>

// Kernel function to be executed on the GPU
__global__ void helloCUDA() {
    printf("Hello, CUDA! This is thread %d\n", threadIdx.x);
}

int main() {
    // Launch the kernel with 10 threads
    helloCUDA << <1, 10 >> > ();

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    std::cout << "GPU kernel execution complete!\n";
    return 0;
}